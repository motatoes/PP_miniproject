#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include <hiprand/hiprand_kernel.h>


//cube libraries

/*ACO parameters*/
  //Number of nodes in the graph
  #define GRAPH_SIZE 32
  //Number of iteration in ACO algorithm
  #define ACO_ITER_MAX 2 
  //evaporation rate
  #define EVAP_RATE 0.3
  //influence rate of the pheromone 
  #define ALPHA 0.8
  //influence rate of the heuristic (distance)
  #define BETA 0.2
  //Initial level of pheromone
  #define INIT_PHEROMONE 5
  //Update pheromone constant
  #define UPDT_PHEROMONE_CONST 2
  //Number of ants
  #define NB_ANT 32 
 /*End ACO parameters*/ 
 
/*GPU parameters*/
  #define GRID_SIZE 1
  #define ITER_BENCHMARK 100
/*End GPU param eters*/

   
//solutions to be used by visualize.cu
int* h_solutions, *d_solutions;
int *h_graph, *d_graph; 


#include "visualize.c"



//random numbers macros
#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)




//function prototypes
void h_datainit_graph(int*, int);
void h_datainit_pheromone(float*, int);
void h_update_pheromone(float* h_pheromone, int size);
float* h_sum_probability(int* h_graph, float* h_pheromone, int size);
void h_update_probability(int* h_graph,float* h_pheromone,float* h_probability, int size, float* h_sum);
int* h_find_best_solution(int* h_solutions, int* h_length, int size);

//a macro function that takes as parameters the indexes
//of a 2d matrix and it's row size, and returns the 
//serialized index
#define SERIALIZE(i,j,row_size) i * row_size + j;


__global__ void ACO_kernel(int* d_graph, float* d_pheromone, float* d_probability, float* d_random_numbers, int* d_solutions,int* d_length)
{
  int tid = threadIdx.x;

  int index,j;
  //initialize the array that contain the solution
  //each thread initialise one row
  for(j=0; j<GRAPH_SIZE ; j++)
  {
    index = SERIALIZE(tid,j,GRAPH_SIZE);
    d_solutions[index]=0;
  }


  __syncthreads();


  //Generate the solution
  float rdm;
  index=SERIALIZE(tid,1,GRAPH_SIZE);
  //For the cube it is going to be loop until NB_STEP is reached or solution found 
  while(d_solutions[index-1] != GRAPH_SIZE-1)
  {
      //select the next node based on the probability
      //take a random number between 0 and 1 with 0 excluded
      rdm=d_random_numbers[index];

      //Probability to select the next node
      float Pnext = 0;

      int j,ip;
      for(j=0; j<GRAPH_SIZE; j++)
      {
          ip = SERIALIZE(d_solutions[index-1], j, GRAPH_SIZE);
          Pnext += d_probability[ip];

          //if the random number is less or equal to
          //the probability to select the next node we select it
          if( rdm <= Pnext )
          {
              d_solutions[index]=j;
              break;
          }
       }

      index++;
  }

  __syncthreads();

  //Calculate the length of the path for each ant
  d_length[tid]=0;
  j=0;
  index=SERIALIZE(tid,j,GRAPH_SIZE)
  while(d_solutions[index] != GRAPH_SIZE-1)
  {
      d_length[tid] += d_graph[d_solutions[index]*GRAPH_SIZE + d_solutions[index+1]];
      j++;
      index=SERIALIZE(tid,j,GRAPH_SIZE);
  }

  //Update the pheromone based on constructed solution
  //Each ant update its own path in the pheromone matrix
  index=SERIALIZE(tid,0,GRAPH_SIZE);
  while(d_solutions[index] != GRAPH_SIZE-1)
    {
        j=SERIALIZE(d_solutions[index],d_solutions[index+1],GRAPH_SIZE);
        d_pheromone[j] += UPDT_PHEROMONE_CONST/d_length[tid];
        index++;
    }

}
  

/*
 * Main program and benchmarking 
 */
int main(int argc, char** argv)
{
  

  // allocate host memory 
  unsigned int nb_node              = GRAPH_SIZE; 
  unsigned int size_graph           = GRAPH_SIZE*GRAPH_SIZE;
  unsigned int mem_size_graph_int   = sizeof(int) * size_graph;
  unsigned int mem_size_graph_float = sizeof(float) * size_graph;
  unsigned int mem_size_ant         = sizeof(int) * NB_ANT;
  unsigned int mem_size_solution    = sizeof(int)*NB_ANT*GRAPH_SIZE;    
  h_graph                           = (int*)malloc(mem_size_graph_int); 
  float* h_pheromone                = (float*)malloc(mem_size_graph_float);
  float* h_probability              = (float*)malloc(mem_size_graph_float);
  //global variable
  h_solutions                       = (int*)malloc(mem_size_solution);
  int*   h_length                   = (int*)malloc(mem_size_ant);

  //Initialise random numbers
  float *d_random_numbers;
  //create hiprand generator object
  hiprandGenerator_t gen;
  /* Allocate n floats on device */
  CUDA_CALL(hipMalloc((void **)&d_random_numbers, NB_ANT * nb_node *sizeof(float)));

  /* Create pseudo-random number generator */
  CURAND_CALL(hiprandCreateGenerator(&gen, 
              HIPRAND_RNG_PSEUDO_DEFAULT));
  /* Set seed */
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
              time(NULL)));
  /* Generate n floats on device */
  CURAND_CALL(hiprandGenerateUniform(gen, d_random_numbers, NB_ANT * nb_node ));

  printf("Input size : %d\n", GRAPH_SIZE);

  //Initialise the graph, the pheromone and the probabilities
  h_datainit_graph(h_graph, nb_node);
  h_datainit_pheromone(h_pheromone, nb_node);
  float* h_sum = h_sum_probability(h_graph, h_pheromone, nb_node);
  h_update_probability(h_graph, h_pheromone, h_probability, nb_node, h_sum);



  // allocate device memory
  // d_graph;
  cutilSafeCall(hipMalloc((void**) &d_graph, mem_size_graph_int));
  float* d_pheromone;
  cutilSafeCall(hipMalloc((void**) &d_pheromone, mem_size_graph_float));
  float* d_probability;
  cutilSafeCall(hipMalloc((void**) &d_probability, mem_size_graph_float));
  //global variable
  d_solutions;
  cutilSafeCall(hipMalloc((void**) &d_solutions, mem_size_solution));

  //Array that contain the length of the path generated by each ant
  int* d_length;
  cutilSafeCall(hipMalloc((void**) &d_length, mem_size_ant));  
  

  // copy host memory to device

  //The graph needs to be copied in the constant memory!!!!!!!!!!!!!!!
  cutilSafeCall(hipMemcpy(d_graph, h_graph, 
              mem_size_graph_int, hipMemcpyHostToDevice));

  cutilSafeCall(hipMemcpy(d_pheromone, h_pheromone, 
              mem_size_graph_float, hipMemcpyHostToDevice));

  cutilSafeCall(hipMemcpy(d_probability, h_probability, 
              mem_size_graph_float, hipMemcpyHostToDevice));             

  // set up kernel for execution
  printf("Run %d Kernels.\n\n", ITER_BENCHMARK);
  unsigned int timer = 0;
  cutilCheckError(cutCreateTimer(&timer));
  cutilCheckError(cutStartTimer(timer));  

int* h_best_solution;
// execute kernel
  //for (int j = 0; j < ITER_BENCHMARK; j++) 
      for(int i = 0; i < ACO_ITER_MAX; i++){

          ACO_kernel<<<1, NB_ANT >>>(d_graph, d_pheromone, d_probability, d_random_numbers, d_solutions, d_length);
          // copy result from device to host
          cutilSafeCall(hipMemcpy(h_solutions, d_solutions, 
               mem_size_solution, hipMemcpyDeviceToHost));
          cutilSafeCall(hipMemcpy(h_length, d_length, 
               mem_size_ant, hipMemcpyDeviceToHost));
          cutilSafeCall(hipMemcpy(h_pheromone, d_pheromone, 
               mem_size_graph_float, hipMemcpyDeviceToHost));
          //find the best solution and its length
          h_best_solution = h_find_best_solution(h_solutions,h_length,NB_ANT);
          //update the pheromone (evaporation)
          h_update_pheromone(h_pheromone,nb_node);
          //update the probability
          h_sum = h_sum_probability(h_graph, h_pheromone, nb_node);
          h_update_probability(h_graph, h_pheromone, h_probability, nb_node, h_sum);
          //copy back the update pheromone and probability to the GPU
          cutilSafeCall(hipMemcpy(d_pheromone, h_pheromone, 
              mem_size_graph_float, hipMemcpyHostToDevice));

          cutilSafeCall(hipMemcpy(d_probability, h_probability, 
              mem_size_graph_float, hipMemcpyHostToDevice));

        //regenerate random numbers
        /* Set seed */
        CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
                    time(NULL)));
        /* Generate n floats on device */
        CURAND_CALL(hiprandGenerateUniform(gen, d_random_numbers, NB_ANT * nb_node ));




        cutilSafeCall(hipMemcpy(h_solutions, d_solutions, sizeof(int) * NB_ANT * GRAPH_SIZE, hipMemcpyDeviceToHost));
        init_visualization(argc,   argv); 
      }

  printf("the best path is: \n");
  int i = 1;
  printf("%d ",h_best_solution[0]);
  while(h_best_solution[i-1] != GRAPH_SIZE-1)
  {
    printf("%d ",h_best_solution[i]);
    i++;
  }
  printf("\n");
  // printf("last set of solutions \n");
  // int index;
  // for(int i=0; i<NB_ANT; i++)
  // {
  //   for(int j=0; j<nb_node; j++)
  //   {
  //       index = SERIALIZE(i,j,nb_node);
  //       printf("%d ",h_solutions[index]);
  //   }       
  //    printf("\n");
  // }

  // printf("last set of length solution \n");
  // for(int i=0; i<NB_ANT; i++)
  // {
  //   printf("%d ",h_length[i]);
  // }
  // printf("\n");

  // check if kernel execution generated and error
  cutilCheckMsg("Kernel execution failed");

  // wait for device to finish
  hipDeviceSynchronize();

  // stop and destroy timer
  cutilCheckError(cutStopTimer(timer));
  double dSeconds = cutGetTimerValue(timer)/(1000.0);
  double dNumOps = (size_graph * 3 + NB_ANT * GRAPH_SIZE + NB_ANT);
  double gflops = dNumOps/dSeconds/1.0e9;

  //Log througput
  printf("Throughput = %.4f GFlop/s\n", gflops);
  printf("Times = %.4f s\n", dSeconds);
  cutilCheckError(cutDeleteTimer(timer));

  // clean up memory
  free(h_graph);
  free(h_pheromone);
  free(h_probability);
  free(h_solutions);
  free(h_length);
  free(h_sum);
  free(h_best_solution);
  cutilSafeCall(hipFree(d_graph));
  cutilSafeCall(hipFree(d_pheromone));
  cutilSafeCall(hipFree(d_probability));
  cutilSafeCall(hipFree(d_solutions));
  cutilSafeCall(hipFree(d_length));

  CURAND_CALL(hiprandDestroyGenerator(gen));
  CUDA_CALL(hipFree(d_random_numbers)); 

  // exit and clean up device status
  hipDeviceReset();
}

// 
void h_datainit_graph(int* h_graph, int size)
{    
    //same method as the CPU version
    int i,j,index;
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);

            if(i < j) {

                h_graph[index] = 1;
            }
            else {
            h_graph[index] = 0;
          }
        }
    }
 
}

void h_datainit_pheromone(float* h_pheromone, int size)
{
  //same method as the CPU version
    int i,j,index;
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);
            if(i < j)
                h_pheromone[index] = INIT_PHEROMONE;
            else{
            h_pheromone[index] = 0;}
        }
    }

}




void h_update_pheromone(float* h_pheromone, int size)
{
    int i,j,index;
    //evaporation
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
          index = SERIALIZE(i,j,size);
            if(h_pheromone !=0)
            {
              h_pheromone[index] = (1-EVAP_RATE) * h_pheromone[index];
            }
        }
    }
}


float* h_sum_probability(int* h_graph, float* h_pheromone, int size)

{
    int i,j,index;
    float* sum = (float*)malloc(sizeof(float)*size);
    for(i=0 ; i<size ; i++)
    {
        sum[i]=0;
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);
            if(h_graph[index] != 0){
                sum[i] += pow(h_pheromone[index],ALPHA) * pow(1/h_graph[index],BETA);
            }
        }
    }
    return sum;
}


void h_update_probability(int* h_graph,float* h_pheromone,float* h_probability, int size, float* h_sum)
{
    //same methode as the CPU version
    int i,j,index;
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);
            if(h_graph[index] != 0)
            {
                h_probability[index] = pow(h_pheromone[index],ALPHA) * pow(1/h_graph[index],BETA)/h_sum[i];
            }
            else{
                h_probability[index] = 0;
            }
        }
    }

}
//
int* h_find_best_solution(int* h_solutions, int* h_length, int size)
{
  //find the shortest length and path
  int* h_best_solution = (int*)malloc(sizeof(int) * GRAPH_SIZE);
  int Lmin=h_length[0];
  int index;
  for(int i=1; i<size; i++)
  {
      if(h_length[i]<=Lmin)
      {
        Lmin = h_length[i];
        index = SERIALIZE(i,0,GRAPH_SIZE);
        memcpy(h_best_solution, &(h_solutions[index]), sizeof(int)*GRAPH_SIZE);
      }   
  }
  return h_best_solution;
}
